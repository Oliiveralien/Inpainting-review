#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/blob.hpp"

namespace caffe {
//sigmoid
template <typename Dtype>
__global__ void caffe_SigmoidForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = 1. / (1. + exp(-in[index]));
  }
}
template <typename Dtype>
__global__ void caffe_SigmoidBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype sigmoid_x = out_data[index];
    out_diff[index] = in_diff[index] * sigmoid_x * (1 - sigmoid_x);
  }
}
template <> 
void caffe_gpu_sigmoid_forward<float>(int N, const float* bottom, float* top) {
  caffe_SigmoidForward<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, bottom, top);
  CUDA_POST_KERNEL_CHECK;
}
template <> 
void caffe_gpu_sigmoid_forward<double>(int N, const double* bottom, double* top) {
  caffe_SigmoidForward<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, bottom, top);
  CUDA_POST_KERNEL_CHECK;
}
template <> 
void caffe_gpu_sigmoid_backward<float>(int N,const float* top_data, const float* top_diff, float* bottom_diff) {
   caffe_SigmoidBackward<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, top_diff, top_data, bottom_diff);
   CUDA_POST_KERNEL_CHECK;
}
template <> 
void caffe_gpu_sigmoid_backward<double>(int N,const double* top_data, const double* top_diff, double* bottom_diff) {
   caffe_SigmoidBackward<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, top_diff, top_data, bottom_diff);
   CUDA_POST_KERNEL_CHECK;
}

//relu
template <typename Dtype>
__global__ void caffe_ReLUForward(const int n, const Dtype* in, Dtype* out,
    Dtype negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : in[index] * negative_slope;
  }
}
template <typename Dtype>
__global__ void caffe_ReLUBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) * negative_slope);
  }
}
template <>
void caffe_gpu_relu_forward<float>(int N, const float* bottom, float* top) {
  caffe_ReLUForward<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, bottom, top, 0);
  CUDA_POST_KERNEL_CHECK;
}
template <>
void caffe_gpu_relu_forward<double>(int N, const double* bottom, double* top) {
  caffe_ReLUForward<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, bottom, top, 0);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_relu_backward<float>(int N, const float* top_data,const float* top_diff, float* bottom_diff) {
    caffe_ReLUBackward<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
        N, top_diff, top_data, bottom_diff, 0);
    CUDA_POST_KERNEL_CHECK;
}
template <>
void caffe_gpu_relu_backward<double>(int N, const double* top_data,const double* top_diff, double* bottom_diff) {
    caffe_ReLUBackward<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
        N, top_diff, top_data, bottom_diff, 0);
    CUDA_POST_KERNEL_CHECK;
}
//tanh
template <typename Dtype>
__global__ void caffe_TanHForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = tanh(in[index]);
  }
}
template <typename Dtype>
__global__ void caffe_TanHBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype tanhx = out_data[index];
    out_diff[index] = in_diff[index] * (1 - tanhx * tanhx);
  }
}
template <>
void caffe_gpu_tanh_forward<float>(int N, const float* bottom, float* top) {
  caffe_TanHForward<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, bottom, top);
  CUDA_POST_KERNEL_CHECK;
}
template <>
void caffe_gpu_tanh_forward<double>(int N, const double* bottom, double* top) {
  caffe_TanHForward<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, bottom, top);
  CUDA_POST_KERNEL_CHECK;
}
template <>
void caffe_gpu_tanh_backward<float>(int N, const float* top_data,const float* top_diff, float* bottom_diff) {
    caffe_TanHBackward<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
        N, top_diff, top_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
}
template <>
void caffe_gpu_tanh_backward<double>(int N, const double* top_data,const double* top_diff, double* bottom_diff) {
    caffe_TanHBackward<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
        N, top_diff, top_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
}

template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <>
void caffe_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));

__global__ void popc_kernel(const int n, const float* a,
    const float* b, uint8_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = __popc(static_cast<uint32_t>(a[index]) ^
                      static_cast<uint32_t>(b[index]));
  }
}

__global__ void popcll_kernel(const int n, const double* a,
    const double* b, uint8_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = __popcll(static_cast<uint64_t>(a[index]) ^
                      static_cast<uint64_t>(b[index]));
  }
}

template <>
uint32_t caffe_gpu_hamming_distance<float>(const int n, const float* x,
                                  const float* y) {
  // TODO: Fix caffe_gpu_hamming_distance (see failing unit test
  // TestHammingDistanceGPU in test_math_functions.cpp).
  NOT_IMPLEMENTED;
  thrust::device_vector<uint8_t> popcounts(n);
  // NOLINT_NEXT_LINE(whitespace/operators)
  popc_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, thrust::raw_pointer_cast(popcounts.data()));
  return thrust::reduce(popcounts.begin(), popcounts.end(),
                        (uint32_t) 0, thrust::plus<uint32_t>());
}

template <>
uint32_t caffe_gpu_hamming_distance<double>(const int n, const double* x,
                                   const double* y) {
  // TODO: Fix caffe_gpu_hamming_distance (see failing unit test
  // TestHammingDistanceGPU in test_math_functions.cpp).
  NOT_IMPLEMENTED;
  thrust::device_vector<uint8_t> popcounts(n);
  // NOLINT_NEXT_LINE(whitespace/operators)
  popcll_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, thrust::raw_pointer_cast(popcounts.data()));
  return thrust::reduce(popcounts.begin(), popcounts.end(),
                        /* NOLINT_NEXT_LINE(build/include_what_you_use) */
                        (uint32_t) 0, thrust::plus<uint32_t>());
}

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

template <typename Dtype>
__global__ void permute_gpu_matrix(const int n,
                                    const int dimsize,
                                    const Dtype* datain, 
                                    Dtype* dataout,
                                    const int *order,
                                    const int * dstdim_capacity,
                                    const int flipdim, 
                                    const int * dstdimensions,                                
                                    const int * dim_capacity) {
    int curidx=0;
    int  idx[4],dstidx[4];
    
    
        CUDA_KERNEL_LOOP(i, n) 
        //for(int i=0;i<count;i++)
        {
            int temp=i;
            for(int j=0;j<dimsize;j++)
            {
                dstidx[j]= temp / dstdim_capacity[j];
                temp = temp % dstdim_capacity[j];
            }
            if(flipdim >= 0)
            {
                dstidx[flipdim] = dstdimensions[flipdim] - 1 - dstidx[flipdim] ;
            }
            for(int j=0;j<dimsize;j++)
            {
                idx[order[j]] = dstidx[j];
            }    
            curidx=0;
            for(int j=0;j<dimsize;j++)
            {    
                    curidx += idx[j]* dim_capacity[j];
            }
            dataout[i] = datain[curidx];
        }
}

template <typename Dtype>
void caffe_gpu_permute(const Dtype * datain, Dtype * dataout, const int * dimensions, const int * order, const int dimsize,const int flipdim)
{  /*
    * implement matlab permute, with additional operation. 
    *1. permutation matrix dimensions 
    *2. flip given dimension
    *liangji, 20150113
   */
    CHECK(dimsize<=4)<<"currently caffe_gpu_permute only support matrix dimensions smaller than 4, got "<<dimsize<<" here.";
    shared_ptr<Blob <int> > dim_capacity_data(new Blob<int>());
    shared_ptr<Blob <int> > dstdimensions_data(new Blob<int>());
    shared_ptr<Blob <int> > dstdim_capacity_data(new Blob<int>());
    shared_ptr<Blob <int> > order_dim_data(new Blob<int>());
   
    dim_capacity_data->Reshape(1,1,1,dimsize);
    dstdimensions_data->Reshape(1,1,1,dimsize);
    dstdim_capacity_data->Reshape(1,1,1,dimsize);
    order_dim_data->Reshape(1,1,1,dimsize);
   
    caffe_set(dimsize, 1, dim_capacity_data->mutable_cpu_data());
    caffe_set(dimsize, 0, dstdimensions_data->mutable_cpu_data());
    caffe_set(dimsize, 1, dstdim_capacity_data->mutable_cpu_data());
    caffe_set(dimsize, 0, order_dim_data->mutable_cpu_data());
    
    caffe_copy(dimsize, order, order_dim_data->mutable_cpu_data());
    
    int  *dim_capacity, *dstdimensions,*dstdim_capacity;//,*order_dim;
    dim_capacity = dim_capacity_data->mutable_cpu_data();
    dstdimensions = dstdimensions_data->mutable_cpu_data();
    dstdim_capacity = dstdim_capacity_data->mutable_cpu_data();
    //order_dim = order_dim_data->mutable_cpu_data();

    for(int i = dimsize-2; i>=0; i-- )
    {
        dim_capacity[i]=1;
        for(int j=i+1;j<dimsize;j++)
            dim_capacity[i] *= dimensions[j];
    }
    for(int i =0; i<dimsize; i++ )
    {
        dstdimensions[i] = dimensions[order[i]];
    }
    for(int i = dimsize-2; i>=0; i-- )
    {
        dstdim_capacity[i]=1;
        for(int j=i+1;j<dimsize;j++)
            dstdim_capacity[i] *= dstdimensions[j];
    }
    
    int count = dim_capacity[0]* dimensions[0];
    permute_gpu_matrix<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
                                                                                    dimsize,
                                                                                    datain, 
                                                                                    dataout,
                                                                                    order_dim_data->gpu_data(),
                                                                                    dstdim_capacity_data->gpu_data(),
                                                                                    flipdim, 
                                                                                    dstdimensions_data->gpu_data(),
                                                                                    dim_capacity_data->gpu_data());
    
    CUDA_POST_KERNEL_CHECK;
    return;
}
template void caffe_gpu_permute<float>(const float * datain, float * dataout, const int * dimensions, const int * order, const int dimsize,const int flipdim);
template void caffe_gpu_permute<double>(const double * datain, double * dataout, const int * dimensions, const int * order, const int dimsize,const int flipdim);


template <typename Dtype>
__global__ void bound_kernel(const int n, const Dtype* a, const Dtype min_val,
    const Dtype max_val, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = min(max(a[index], min_val), max_val);
  }
}
template <typename Dtype>
void caffe_gpu_bound(const int N, const Dtype* a, const Dtype min_val, 
    const Dtype max_val, Dtype* y) {
  bound_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, min_val, max_val, y);
}
template void caffe_gpu_bound<float>(const int N, const float* a, const float min_val, const float max_val, float* y);
template void caffe_gpu_bound<double>(const int N, const double* a, const double min_val, const double max_val, double* y);


}  // namespace caffe
